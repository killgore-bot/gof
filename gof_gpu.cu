// Rohit Khiani
// id - khianirm
// CS-632
// hw-5


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16


__global__ void rules(int size, int *simulation, int *newsimulation)
{
    // We want id ∈ [1,size]
    int d_row = blockDim.y * blockIdx.y + threadIdx.y + 1;
    int d_col = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int id = d_row * (size+2) + d_col;

    int count;

    if (d_row <= size && d_col <= size) {

        // Get the number of neighbors for a given simulation point
        count = simulation[id+(size+2)] + simulation[id-(size+2)] //upper lower
                     + simulation[id+1] + simulation[id-1]             //right left
                     + simulation[id+(size+3)] + simulation[id-(size+3)] //diagonals
                     + simulation[id-(size+1)] + simulation[id+(size+1)];

        int cell = simulation[id];
        // Here we have explicitly all of the game rules
        if (cell == 1 && count < 2)
            newsimulation[id] = 0;
        else if (cell == 1 && (count == 2 || count == 3))
            newsimulation[id] = 1;
        else if (cell == 1 && count > 3)
            newsimulation[id] = 0;
        else if (cell == 0 && count == 3)
            newsimulation[id] = 1;
        else
            newsimulation[id] = cell;
    }
}

void errorCheck()
{
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}

int main(int argc, char* argv[])
{
    int i,j;
    int* matrix;
    int* d_matrix;
    int* t_matrix;
    int* d_tmpsimulation;
    int size = 4999;
    int maxIter = 1<<12;

    int N = sizeof(int)*(size+2)*(size+2);
    matrix = (int*)malloc(N);


    hipMalloc(&d_matrix, N);
    hipMalloc(&t_matrix, N);


    for(i = 0; i<=size+1; i++) {
        for(j = 0; j<=size+1; j++) {
          if (i == 0 || j == size ){
              matrix[i*(size+2)+j] = 0;
          }
          else if (i == size || j== size){
              matrix[i*(size+2)+j] = 0;
          }
            else matrix[i*(size+2)+j] = rand() % 2;
        }
    }


    hipMemcpy(d_matrix, matrix, N, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE,1);
    int linsimulation = (int)ceil(size/(float)BLOCK_SIZE);
    dim3 simulationSize(linsimulation,linsimulation,1);

    dim3 cpyBlockSize(BLOCK_SIZE,1,1);
    dim3 cpysimulationRowssimulationSize((int)ceil(size/(float)cpyBlockSize.x),1,1);
    dim3 cpysimulationColssimulationSize((int)ceil((size+2)/(float)cpyBlockSize.x),1,1);

    errorCheck();

    for (i = 0; i<maxIter; i++) {
        rules<<<simulationSize, blockSize>>>(size, d_matrix, t_matrix);

        d_tmpsimulation = d_matrix;
        d_matrix = t_matrix;
        t_matrix = d_tmpsimulation;
    }

    for (i = 4096; i< 5000; i++) {


        rules<<<simulationSize, blockSize>>>(size, d_matrix, t_matrix);


        d_tmpsimulation = d_matrix;
        d_matrix = t_matrix;
        t_matrix = d_tmpsimulation;
    }


    hipMemcpy(matrix, d_matrix, N, hipMemcpyDeviceToHost);


    for (i = 0; i<=size+1; i++) {
        for (j = 0; j<=size+1; j++) {
            printf("%d",matrix[i*(size+2) + j]);
        }
        printf("\n");
    }



    hipFree(d_matrix);
    hipFree(t_matrix);
    free(matrix);

    return 0;
}
